#include "hip/hip_runtime.h"
#include "KMeans.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>


#define TILE_SIZE 32   //start the block size.
#define MAX_BLOCKS_PER_GRID 65535   //Start the max grid size.

__constant__ Vector2 clustersconst[3];

/************************************************************************/
//
// GPU kernel for Kmean clustering.
//
/************************************************************************/
__global__ void KMeansKernelGPU( Datapoint* data, long n, int k )
{
	long blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	long threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x; //calculate the threadId in grid. 
	if(threadId < n){			//check the condition. 
		data[threadId].altered = false;				// reset every element with altered 
		for(int clusterNo=0; clusterNo<k; clusterNo++)				//Loop over all the cluster
		{
			if(clustersconst[clusterNo].distSq(data[threadId].p) < clustersconst[data[threadId].cluster].distSq(data[threadId].p)){			//Check for distance with all the clusters.
				if(data[threadId].cluster != clusterNo){				//if elemets need to change the cluster.
					data[threadId].cluster = clusterNo;
					data[threadId].altered = true;						//change the altered value.
				}
			}
		}
	}
}


/************************************************************************/
// 
//	KmeanGPU method for kernel implementation of Kmean. 
//
/************************************************************************/
bool KMeansGPU( Datapoint* data, long n, Vector2* clusters, int k ){

	// Error return value
	hipError_t status;						//Status variable for cuda call.
	long noElements =  n * sizeof(Datapoint);		//Total number of elements.
	Datapoint *dataDevice;							// Datapoints variable for device.
	status = hipMalloc((void**) &dataDevice, noElements);		//Cuda melloc
	if (status != hipSuccess)									//Error check
	{
		std::cout << "Kernel failed (Ad alloc): " << hipGetErrorString(status) << 
					 std::endl;
		return false;
	}

	dim3 dimBlock(TILE_SIZE, 1);			//Initialize blocks.
	int gridx = 1;							//Vaiable for Grid size.
	int gridy = 1;							//Vaiable for Grid size
	//Logic to initialize 1D or 2D grids according to number of elements passed.
	if(n/TILE_SIZE < MAX_BLOCKS_PER_GRID+1)
		gridx = ceil((float)n/TILE_SIZE);
	else{
		gridx = MAX_BLOCKS_PER_GRID;
		gridy = ceil((float)n/(TILE_SIZE*MAX_BLOCKS_PER_GRID));
	}
	dim3 dimGrid(gridx, gridy);	 //Initialize grid size.
	bool is_KMean_done = true;			 //is_KMean_done to stop kmean clustring algo.
	
	while(is_KMean_done){
		is_KMean_done=false;			//initialize flag to false.
		status = hipMemcpy(dataDevice, data, noElements, hipMemcpyHostToDevice);			//Memcpy for data elements.
		if (status != hipSuccess)					//Error check
		{
			std::cout << "Kernel failed (data Memcpy): " << hipGetErrorString(status) << 
							std::endl;
			hipFree(dataDevice);
			return false;
		}
	
		status = hipMemcpyToSymbol(HIP_SYMBOL(clustersconst), clusters, k*sizeof(Vector2), 0, hipMemcpyHostToDevice);				//Copying cuda constant memory.
		if (status != hipSuccess)																						//Error checking
		{
			std::cout << "Constant memory copy failed in const memory: " << hipGetErrorString(status) << 
						 std::endl;
			hipFree(dataDevice);
			return false;
		}

		KMeansKernelGPU<<<dimGrid, dimBlock>>>(dataDevice, n, k);				//Launching kernel
		// Wait for completion
		hipDeviceSynchronize();		

		// Check for errors
		status = hipGetLastError();
		if (status != hipSuccess)
		{
			std::cout << "Kernel failed on execution: " << hipGetErrorString(status) << std::endl;
			hipFree(dataDevice);
			return false;
		}
		
		status = hipMemcpy(data, dataDevice, noElements, hipMemcpyDeviceToHost);		//memcpy from device to host back the data points.
		if (status != hipSuccess)														//Error check
		{
			std::cout << "Kernel failed (data Memcpy) hipMemcpyDeviceToHost: " << hipGetErrorString(status) << 
							std::endl;
			hipFree(dataDevice);
			return false;
		}

		// Logic to update cluster.	
		for (int clusterNo=0; clusterNo<k; clusterNo++){						//loop over cluster.
			long numElements=0;													//initalize number of elements.
			for(long element=0; element<n; element++)							//Loop over elements.
			{
				if(data[element].cluster == clusterNo){							//check the data elements for cluster and update if needed.
					clusters[clusterNo].x += data[element].p.x;
					clusters[clusterNo].y += data[element].p.y;
					numElements++;
				}
				if(data[element].altered==true){								//Check for altered elements and update flag if nothing is updated.
					is_KMean_done=true;
				}
			}
			if(numElements>0){												//Update the mean to cluster point.
				clusters[clusterNo].x /= numElements;
				clusters[clusterNo].y /= numElements;
			}
		}
	}

	hipFree(dataDevice);					//Free cuda mememory
	return true;
}
